#include "hip/hip_runtime.h"
// cuda_simulation.cu
#include "cuda_simulation.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

// CUDA kernel for vector addition
__global__ void vecAdd(const float* A, const float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
        C[i] = A[i] + B[i];
}

void runCUDASimulation() {
    const int n = 1024;
    size_t size = n * sizeof(float);
    std::vector<float> h_A(n, 1.0f), h_B(n, 2.0f), h_C(n, 0.0f);
    
    float *d_A, *d_B, *d_C;
    hipError_t err;
    
    // Allocate device memory
    err = hipMalloc((void**)&d_A, size);
    if(err != hipSuccess) { std::cerr << "hipMalloc failed for d_A\n"; return; }
    err = hipMalloc((void**)&d_B, size);
    if(err != hipSuccess) { std::cerr << "hipMalloc failed for d_B\n"; hipFree(d_A); return; }
    err = hipMalloc((void**)&d_C, size);
    if(err != hipSuccess) { std::cerr << "hipMalloc failed for d_C\n"; hipFree(d_A); hipFree(d_B); return; }
    
    // Copy input data to device
    err = hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    if(err != hipSuccess) { std::cerr << "hipMemcpy failed for d_A\n"; return; }
    err = hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);
    if(err != hipSuccess) { std::cerr << "hipMemcpy failed for d_B\n"; return; }
    
    // Launch kernel: choose block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    auto start = std::chrono::high_resolution_clock::now();
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> execTime = end - start;
    std::cout << "CUDA kernel execution time: " << execTime.count() << " ms\n";
    
    // Copy result back to host
    err = hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);
    if(err != hipSuccess) { std::cerr << "hipMemcpy failed for result\n"; }
    
    // Check results (optional)
    bool success = true;
    for (int i = 0; i < n; i++) {
        if(h_C[i] != 3.0f) { success = false; break; }
    }
    std::cout << "CUDA Simulation: " << (success ? "Success!" : "Failure!") << std::endl;
    
    // Clean up device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}